#include "hip/hip_runtime.h"
﻿#include <cstdlib>
#include <ostream>
#include <iostream>
#include <iomanip>
#include <chrono>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include ""
#include <math.h>
#include <windows.h>

#define Mbig 27700
#define Nbig 12300
#define BLOCK_SIZE 16
#define M 2570
#define N 2270

using namespace std;

void checkOnError(hipError_t cudaStatus) {
	if (cudaStatus != hipSuccess) {
		cout << "CUDA return error code: " << cudaStatus;
		cout << " " << hipGetErrorString(cudaStatus) << endl;
	}
}

void randomElements(short* matrix, long long matrixRows, long long matrixCols) {
	srand(time(NULL));
	for (long long i = 0; i < matrixRows; i++) {
		for (long long j = 0; j < matrixCols; j++) 
			matrix[i * matrixCols + j] = rand() % 100 + 1;
	}
}

void showMatrix(short* matrix, int matrixRows, int matrixCols) {
	for (int i = 0; i < matrixRows; i++) {
		for (int j = 0; j < matrixCols; j++)
			cout << setw(4) << matrix[i * matrixCols + j];
		cout << '\n';
	}
}

__global__ void kernel(short* sourseMatrix, int sourseMatrixRow, int sourseMatrixCol, short* resultMatrix) {
	int rows = blockIdx.y * blockDim.y + threadIdx.y;
	int cols = blockIdx.x * blockDim.x + threadIdx.x;
	if ((rows <= sourseMatrixRow) && (cols <= sourseMatrixCol)) {
		resultMatrix[(rows * sourseMatrixCol + cols) * 2] = sourseMatrix[rows * sourseMatrixCol + cols];
		__syncthreads();
		resultMatrix[(rows * sourseMatrixCol + cols) * 2 + 1] = sourseMatrix[rows * sourseMatrixCol + cols];
		__syncthreads();
	}
}

void transformMatrixGPU(short* matrix_In, int rows, int cols, short* result) {
	hipEvent_t start;
	hipEvent_t stop;
	short* matIn;
	short* matOut;
	float time;
	checkOnError(hipMalloc((void**)&matIn, rows * cols * sizeof(short)));
	checkOnError(hipMemcpy(matIn, matrix_In, rows * cols * sizeof(short), hipMemcpyHostToDevice));
	checkOnError(hipMalloc((void**)&matOut, rows * cols * 2 * sizeof(short)));
	dim3 block(16, 4);
	dim3 grid(cols / block.x, rows / block.y);
	if (cols % block.x != 0) grid.x++;
	if (rows % block.y != 0) grid.y++;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	kernel << <grid, block >> > (
		matIn, 
		rows,
		cols, 
		matOut);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "GPU time: " << time << " ms" << endl;
	hipMemcpy(result, matOut, rows * cols * 2 * sizeof(short),
		hipMemcpyDeviceToHost);
	checkOnError(hipFree(matIn));
	checkOnError(hipFree(matOut));
}

bool compareMatrix(short* matrix1, short* matrix2, int matrixRow, int matrixCol) {
	for (auto i = 0; i < matrixRow; i++)
		for (auto j = 0; j < matrixCol; j++)
			if (matrix1[i * matrixCol + j] != matrix2[i * matrixCol + j])
				return true;
	return false;
}

__global__ void kernelSharedGpu(short* first_matrix, int first_matrix_height, int first_matrix_width, short* second_matrix, const int second_matrix_height, const int second_matrix_width) {
	int xIndex = blockIdx.x * 8 + threadIdx.x;
	int yIndex = blockIdx.y * 8 + threadIdx.y;
	int idx = yIndex * first_matrix_width + xIndex;
	int index_out = (yIndex * first_matrix_width + xIndex) * 2;
	__shared__ short block[8][8 * 2];
	if ((xIndex <= first_matrix_width) && (yIndex <= first_matrix_height))
	{
		block[threadIdx.y][threadIdx.x * 2] = first_matrix[idx]; 
		__syncthreads();
		second_matrix[index_out] = block[threadIdx.y][threadIdx.x * 2];
		second_matrix[index_out + 1] = block[threadIdx.y][threadIdx.x * 2];
	}
}

float transform_matrix_gpu_shared(short* first_matrix, const int first_matrix_height, const int first_matrix_width, short* second_matrix, const int second_matrix_height, const int second_matrix_width) {
	hipEvent_t startTime;
	hipEvent_t stopTime;
	short* gpu_first_matrix;
	short* gpu_second_matrix;
	size_t pitch;

	checkOnError(hipMalloc((void**)&gpu_first_matrix, M*N*sizeof(short)));
	checkOnError(hipMemcpy(gpu_first_matrix, first_matrix, M * N * sizeof(short), hipMemcpyHostToDevice));
	checkOnError(hipMalloc((void**)&gpu_second_matrix, M * 2 * N * sizeof(short)));

	dim3 block(8, 8);
	dim3 grid;

	grid.x = first_matrix_width / block.x;
	if (first_matrix_width % block.x != 0) grid.x += 1;

	grid.y = first_matrix_height / block.y;
	if (first_matrix_height % block.y != 0) grid.y += 1;

	checkOnError(hipEventCreate(&startTime));
	checkOnError(hipEventCreate(&stopTime));
	checkOnError(hipEventRecord(startTime));

	kernelSharedGpu << <grid, block >> > (
		gpu_first_matrix,	
		first_matrix_height, 
		first_matrix_width, 
		gpu_second_matrix,
		second_matrix_height,
		second_matrix_width);

	checkOnError(hipEventRecord(stopTime));
	checkOnError(hipEventSynchronize(stopTime));
	float result_time;
	checkOnError(hipEventElapsedTime(&result_time, startTime, stopTime));
	cout << "Shared GPU time: " << result_time << " ms" << endl;
	hipMemcpy(second_matrix, gpu_second_matrix,
		second_matrix_height * second_matrix_width * sizeof(short),
		hipMemcpyDeviceToHost);

	return result_time;
	checkOnError(hipFree(gpu_first_matrix));
	checkOnError(hipFree(gpu_second_matrix));
}

void transformBigMatrixGPU(short* sourseMatrix, int sourseMatrixRow, int sourseMatrixCol, short* resultMatrix) {
	int PARTS = M / 2;
	if (M % 2 != 0) PARTS++;
	short** arrayOfMatrices;
	arrayOfMatrices = (short**)malloc(PARTS * sizeof(short*));
	int i = 0;
	for (int k = 0; k < PARTS; k++)
	{
		arrayOfMatrices[k] = (short*)malloc(sourseMatrixCol * 2 * sizeof(short));
		for (int j = 0; j < N; j++)
		{
		arrayOfMatrices[k][0 * sourseMatrixCol + j] = sourseMatrix[i * sourseMatrixCol + j];
		arrayOfMatrices[k][1 * sourseMatrixCol + j] = sourseMatrix[(i + 1) * sourseMatrixCol + j];
		}
		i += 2;
	}
	cout << '\n';
	short** resultmatrix;
	resultmatrix = (short**)malloc(PARTS * sizeof(short*));
	short* sourseMatrixGPU;
	short* resultMatrixGPU;
	float timeCounter;
	float resultTime = 0;
	hipEvent_t startTime;
	hipEvent_t stopTime;
	for (int partsCounter = 0; partsCounter < PARTS; partsCounter++) {
		checkOnError(hipMalloc((void**)&sourseMatrixGPU, 2 * sourseMatrixCol * sizeof(short)));
		checkOnError(hipMalloc((void**)&resultMatrixGPU, 2 * sourseMatrixCol * 2 * sizeof(short)));
		checkOnError(hipMemcpy(sourseMatrixGPU, arrayOfMatrices[partsCounter], 2 * sourseMatrixCol * sizeof(short), hipMemcpyHostToDevice));
		dim3 block(16, 8);
		dim3 grid(sourseMatrixCol / block.x, sourseMatrixRow / block.y);
		if (sourseMatrixRow % block.y != 0) 
			grid.y++;
		if (sourseMatrixCol % block.x != 0) 
			grid.x++;
		hipEventCreate(&startTime);
		hipEventCreate(&stopTime);
		hipEventRecord(startTime);
		kernel << <grid, block >> > (sourseMatrixGPU, 2, sourseMatrixCol, resultMatrixGPU);
		hipEventRecord(stopTime);
		hipEventSynchronize(stopTime);
		hipEventElapsedTime(&timeCounter, startTime, stopTime);
		resultTime += timeCounter;
		resultmatrix[partsCounter] = (short*)malloc(2 * 2 * sourseMatrixCol * sizeof(short));
		checkOnError(hipMemcpy(resultmatrix[partsCounter], resultMatrixGPU, 2 * 2 * sourseMatrixCol * sizeof(short),
			hipMemcpyDeviceToHost));
		checkOnError(hipFree(sourseMatrixGPU));
		checkOnError(hipFree(resultMatrixGPU));
	}
	cout << "GPU time: " << resultTime << " ms" << endl;
	i = 0;
	for (int k = 0; k < PARTS; k++)
	{

		for (int j = 0; j < N * 2; j++)
		{
			resultMatrix[i * sourseMatrixCol *2 + j]  = resultmatrix[k][0 * sourseMatrixCol * 2 + j];
			resultMatrix[(i + 1) * sourseMatrixCol*2 + j] = resultmatrix[k][1 * sourseMatrixCol * 2 + j];
		}
		i += 2;
	}
}


int main(int argc, char *argv[])
{
#pragma region init
	short* matrix_In;
	short* matrix_In_gpu;
	short* matrix_Out_Cpu;
	short* matrix_Out_Gpu;
	short* matrix_Out_Gpu_big;
	short* matrix_Out_Gpu_shared;
	matrix_In = (short*)malloc(M * N * sizeof(short)+1);
	randomElements(matrix_In, M, N);
	matrix_Out_Cpu = (short*)malloc(M * N * 2 * sizeof(short));
	matrix_Out_Gpu = (short*)malloc(M * N * 2 * sizeof(short));
	matrix_Out_Gpu_shared = (short*)malloc(M * N * 2 * sizeof(short));
	matrix_Out_Gpu_big = (short*)malloc(M * N * 2 * sizeof(short));
	int k = 0;
	auto start_cpu = chrono::steady_clock::now();
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++)
		{
			matrix_Out_Cpu[(i * N + j) * 2] = matrix_In[i * N + j];
			matrix_Out_Cpu[(i * N + j) * 2 + 1] = matrix_In[i * N + j];
		}
	}
	auto end_cpu = chrono::steady_clock::now();
	cout << "CPU time: " << chrono::duration <double, milli>(end_cpu - start_cpu).count() << " ms" << endl;
#pragma endregion

	transformMatrixGPU(matrix_In, M, N, matrix_Out_Gpu);
	transform_matrix_gpu_shared(matrix_In, M, N, matrix_Out_Gpu_shared, M, N * 2);
	//transformBigMatrixGPU(matrix_In, M, N, matrix_Out_Gpu_big);

#pragma region output
	cout << "Matrix in\n";
	showMatrix(matrix_In, 10, 10);
	cout << '\n';
	cout << '\n';
	cout << "Matrix out cpu\n";
	showMatrix(matrix_Out_Cpu, 10, 20);
	cout << '\n';
	cout << '\n';
	cout << "Matrix out gpu simple\n";
	showMatrix(matrix_Out_Gpu, 10, 20);
	cout << '\n';
	cout << '\n';
	cout << "Matrix out gpu shared\n";
	showMatrix(matrix_Out_Gpu_shared, 10, 20);
	cout << '\n';
	cout << '\n';
	cout << "Matrix out gpu big\n";
	showMatrix(matrix_Out_Gpu_big, 10, 20);
	if (!compareMatrix(matrix_Out_Cpu, matrix_Out_Gpu, M, N * 2))
		cout << "Matrix cpu and gpu are the same" << endl; else cout << "MATRIX CPU AND GPU NOT EQUAL" << endl;
	if (!compareMatrix(matrix_Out_Cpu, matrix_Out_Gpu_shared, M, N * 2))
		cout << "Matrix gpu and gpu_shared are the same" << endl; else cout << "MATRIX CPU AND GPU SHARED NOT EQUAL" << endl;
	if (!compareMatrix(matrix_Out_Cpu, matrix_Out_Gpu_big, M, N * 2))
		cout << "Matrix gpu and gpu_big are the same" << endl; else cout << "MATRIX CPU AND GPU BIG NOT EQUAL" << endl;
	checkOnError(hipDeviceReset());
#pragma endregion
	system("pause");
    return 0;
}